#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// Add two float2
__host__ __device__ __forceinline__ float2 operator+(float2 a, float2 b) 
{ 
	a.x += b.x;
	a.y += b.y;
	return a;
}

// Multiply a float2 and a float
__host__ __device__ __forceinline__ float2 operator*(float a, float2 b)
{
	b.x *= a;
	b.y *= a;
	return b;
}

// a structure of 2d points
class Points
{
public:

	// constructor
	__host__ __device__ Points(float* x = nullptr, float* y = nullptr) : m_x(x), m_y(y) {}

	// get a point
	__host__ __device__ __forceinline__ float2 get_point(int idx) const { return make_float2(m_x[idx], m_y[idx]); }

	// set a point
	__host__ __device__ __forceinline__ void set_point(int idx, const float2& p) { m_x[idx] = p.x; m_y[idx] = p.y; }

	// set the points pointers
	__host__ __device__ __forceinline__ void set(float* x, float* y) { m_x = x; m_y = y; }

private:

	// coordinates
	float* m_x;
	float* m_y;
};

// a 2D bounding box
class Bounding_box
{
public:

	// constructor
	__host__ __device__ Bounding_box() : m_p_min(make_float2(0, 0)), m_p_max(make_float2(1.0f, 1.0f)) {}

	// compute the center of the box
	__host__ __device__ float2 computeCenter() const { return 0.5f*(m_p_min + m_p_max); }

	// Get the corner points
	__host__ __device__ __forceinline__ const float2& get_max() const { return m_p_max; }
	__host__ __device__ __forceinline__ const float2& get_min() const { return m_p_min; }

	// Does the box contains the input point?
	__host__ __device__ bool contains(const float2& p) const 
	{ 
		return (p.x >= m_p_min.x) && (p.x < m_p_max.x)
			&& (p.y >= m_p_min.y) && (p.y < m_p_max.y);
	}

	// Define the boundinx box
	__host__ __device__ void set(float min_x, float min_y, float max_x, float max_y)
	{
		m_p_min.x = min_x;
		m_p_min.y = min_y;
		m_p_max.x = max_x;
		m_p_max.y = max_y;
	}

private:

	// corners
	float2 m_p_min;
	float2 m_p_max;
};

// A node in the quadtree
class Quadtree_node
{
public:

	// constructor
	__host__ __device__ Quadtree_node() : m_id(0), m_begin(0), m_end(0) {}

	// get the node id
	__host__ __device__ int id() const { return m_id; }

	// set the node id
	__host__ __device__ void set_id(int new_id) { m_id = new_id; }

	// get the bounding box
	__host__ __device__ __forceinline__ const Bounding_box& bounding_box() const { return m_bounding_box; }

	// set the bounding box
	__host__ __device__ __forceinline__ void set_bounding_box(float min_x, float min_y, float max_x, float max_y) { m_bounding_box.set(min_x, min_y, max_x, max_y); }

	// total number of points in the tree node
	__host__ __device__ __forceinline__ int num_points() const { return m_end - m_begin; }

	// the range of points in the tree node
	__host__ __device__ __forceinline__ int points_begin() const { return m_begin; }
	__host__ __device__ __forceinline__ int points_end()   const { return m_end; }

	// define the range of the tree node
	__host__ __device__ __forceinline__ void set_range(int begin, int end) 
	{ 
		m_begin = begin; 
		m_end = end; 
	}

private:

	// identifier
	int m_id;

	// bounding box of the node
	Bounding_box m_bounding_box;

	// the range of points
	int m_begin, m_end;
};

int main(int argc, char* argv[])
{
    // Query GPU properties
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    cout << "---------------------------------------------" << endl;
    cout << "               GPU PROPERTIES                " << endl;
    cout << "---------------------------------------------" << endl;
    cout << "Device Name: " << dev_prop.name << endl;
    cout << "Memory Clock Rate: " << dev_prop.memoryClockRate/1.0e6 <<  " GHz" << endl;
    cout << "Memory Bandwidth: " << 2.0*dev_prop.memoryClockRate*(dev_prop.memoryBusWidth/8)/1.0e6 <<  " GB/s" << endl;
    cout << "Number of SM: " << dev_prop.multiProcessorCount << endl;
    cout << "Max Threads per SM: " << dev_prop.maxThreadsPerMultiProcessor << endl;
    cout << "Registers per Block: " << dev_prop.regsPerBlock << endl;
    cout << "Shared Memory per Block: " << dev_prop.sharedMemPerBlock << " B" << endl;
    cout << "Total Global Memory per Block: " << dev_prop.totalGlobalMem/1.0e9 << " GB" << endl;
    cout << endl;




    cout << "Closing..." << endl;

    return 0;
}

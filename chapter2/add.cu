#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;

// Add A and B vector on the GPU. Results stored into C
__global__
void addKernel(int n, float* A, float* B, float* C)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < n) C[i] = A[i] + B[i];
}

// Add A and B vector. Results stored into C
int add(int n, float* h_A, float* h_B, float* h_C)
{
  int size = n*sizeof(float);

  // Allocate memory on device and copy data
  float* d_A;
  hipMalloc((void**)&d_A, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  float* d_B;
  hipMalloc((void**)&d_B, size);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  float* d_C;
  hipMalloc((void**)&d_C, size);

  // launch Kernel
  cout << "Running 256 threads on " << ceil(n/256.0f) << " blocks -> " << 256*ceil(n/256.0f) << endl;
  addKernel<<<ceil(n/256.0f),256>>>(n, d_A, d_B, d_C);

  // Transfer results back to host
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}

// C = A + B on a GPU, where A is a vector of 1.0f and B a vector of 2.0f
// The main function takes one argument, the size of the vectors
int main(int argc, char* argv[])
{
  int n = atoi(argv[1]);

  vector<float> h_A(n, 1.0f);
  vector<float> h_B(n, 2.0f);
  vector<float> h_C(n);

  add(n, h_A.data(), h_B.data(), h_C.data());

  for(auto& c : h_C) {
    if(fabs(c-3.0f) > 0.00001f) {
      cout << "Error!" << endl;
      return 1;
    }
  }

  cout << "The program completed successfully" << endl;

  return 0;
}

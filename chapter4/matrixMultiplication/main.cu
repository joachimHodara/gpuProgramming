#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define TILE_WIDTH 16

/**
 * C = A * B 
 */
__global__
void matMul(float* C, const float* A, const float* B, int dim)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < dim && row < dim) {
        float prod = 0.0f;
        for(int i = 0; i < dim; i++)
            prod += A[row*dim+i]*B[i*dim+col];
        C[row*dim+col] = prod;
    }
}

/**
 * C = A * B (tiled)
 */
__global__
void matMulTiled(float* C, const float* A, const float* B, int dim)
{
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int tx = threadIdx.x;
    int by = blockIdx.y;  int ty = threadIdx.y;

    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;

    // Loop over the tiles required to compute the element
    float prod = 0.0f;
    for(int ph = 0; ph < ceil(dim/(float)TILE_WIDTH); ++ph) {

        // 1. Load the tiles into shared memory
        if((row < dim) && (ph*TILE_WIDTH + tx < dim))
            As[ty][tx] = A[row*dim + ph*TILE_WIDTH + tx];
        if((ph*TILE_WIDTH + ty < dim) && (col < dim))
            Bs[ty][tx] = B[(ph*TILE_WIDTH + ty)*dim + col];
        __syncthreads();

        // 2. Dot product
        for(int i = 0; i < TILE_WIDTH; ++i)
            prod += As[ty][i]*Bs[i][tx];
        __syncthreads();
    }

    // 3. Write result
    if((row < dim) && (col < dim)) C[row*dim+col] = prod;
}

int main(int argc, char* argv[])
{
    // Query GPU properties
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    cout << "---------------------------------------------" << endl;
    cout << "               GPU PROPERTIES                " << endl;
    cout << "---------------------------------------------" << endl;
    cout << "Device Name: " << dev_prop.name << endl;
    cout << "Memory Clock Rate: " << dev_prop.memoryClockRate/1.0e6 <<  " GHz" << endl;
    cout << "Memory Bandwidth: " << 2.0*dev_prop.memoryClockRate*(dev_prop.memoryBusWidth/8)/1.0e6 <<  " GB/s" << endl;
    cout << "Number of SM: " << dev_prop.multiProcessorCount << endl;
    cout << "Max Threads per SM: " << dev_prop.maxThreadsPerMultiProcessor << endl;
    cout << "Registers per Block: " << dev_prop.regsPerBlock << endl;
    cout << "Shared Memory per Block: " << dev_prop.sharedMemPerBlock << " B" << endl;
    cout << "Total Global Memory per Block: " << dev_prop.totalGlobalMem/1.0e9 << " GB" << endl;
    cout << endl;

    int dim = atoi(argv[1]);
    int size = dim*dim;

    float sharedMemPerBlock = 2*TILE_WIDTH*TILE_WIDTH*4;
    cout << "shared memory per block: " << sharedMemPerBlock << " B" << endl;
    cout << "can run at most " << int(dev_prop.sharedMemPerBlock/sharedMemPerBlock) << " blocks" << endl;

    // creating matrices on host side
    float* h_A = new float[size];
    float* h_B = new float[size];
    for (int i = 0; i < size; ++i) {
        h_A[i] = 3.0f;
        h_B[i] = 0.0f;
    }
    for (int i = 0; i < size; i+=dim+1)
        h_B[i] = 1.0f;

    // Copy matrices on device side
    float* d_A;
    hipMalloc((void**)&d_A, size*sizeof(float));
    hipMemcpy((void*)d_A, (void*)h_A, size*sizeof(float), hipMemcpyHostToDevice);
    float* d_B;
    hipMalloc((void**)&d_B, size*sizeof(float));
    hipMemcpy((void*)d_B, (void*)h_B, size*sizeof(float), hipMemcpyHostToDevice);

    // Allocate C matrix on device
    float* d_C;
    hipMalloc((void**)&d_C, size*sizeof(float));

    // call Kernel
    int type = atoi(argv[2]);
    if (type == 1) { // "regular" matrix multiplication
        dim3 dimGrid(ceil(dim/16.0f), ceil(dim/16.0f), 1);
        dim3 dimBlock(16, 16, 1);
        matMul<<<dimGrid, dimBlock>>> (d_C, d_A, d_B, dim);
    }
    else if (type == 2) { // "tiled" matrix multiplication
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
        dim3 dimGrid(ceil(dim/dimBlock.x), dim/dimBlock.y, 1);
        matMulTiled<<<dimGrid, dimBlock>>> (d_C, d_A, d_B, dim);
    }
    else
        cout << "invalid argument!" << endl;

    // Recover C matrix from device to host
    float* h_C = new float[size];
    hipMemcpy((void*)h_C, (void*)d_C, size*sizeof(float), hipMemcpyDeviceToHost);

    // Check results
    for (int i = 0; i < size; ++i) {
        if (fabs(h_C[i] - 3.0f) > 0.0001f) {
            cout << "ERROR: something is not right." << endl;
            break;
        }
    }

    // Finalize storage
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete [] h_A;
    delete [] h_B;
    delete [] h_C;

    cout << "Closing..." << endl;

    return 0;
}
